#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <math.h>
extern "C" {
#include "stepper.cuh"
#include "stepper_base.h"
#include "shallow2d.cuh"
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void print_array(float* array, int len) {
	for(int i = 0; i < len; i++) {
	    printf("%.2f ", array[i]);    
	}
	printf("\n");
}

int main(int argc, char** argv){
	int nx = 3, ny = 3, ng = 1, nfield = 1;
	int nx_all = nx + 2*ng;
    int ny_all = ny + 2*ng;
	int nc = nx_all * ny_all;
    int N  = nfield * nc;
  	float* u  = (float*) malloc((4*N + 6*nx_all)* sizeof(float));
  	float* u_ture  = (float*) malloc((4*N + 6*nx_all)* sizeof(float));
    float* v  = u + N;
    float* f  = u + 2*N;
    float* g  = u + 3*N;
    float* scratch = u + 4*N;
    srand(time(NULL));
    // set
    int i;
    for (i = 0; i < 4*N + 6*nx_all; i++) {
    	u[i] = cos((float)i/float(4*N + 6*nx_all));
    }
    float dtcdx2 = 0.3, dtcdy2 = 0.3;

	central2d_predict_base(v, scratch, u, f, g, dtcdx2, dtcdy2,
                  nx_all, ny_all, nfield);

	// baseline result
	for (i = 0; i < 4*N + 6*nx_all; i++) {
    	u_ture[i] = u[i];
    }

	// reset
	printf("Test linearized series code. \n");
    for (i = 0; i < 4*N + 6*nx_all; i++) {
    	u[i] = cos((float)i/float(4*N + 6*nx_all));
    }

	central2d_predict_base_linear(v, scratch, u, f, g, dtcdx2, dtcdy2,
              nx_all, ny_all, nfield);

	printf("Check correctness\n");
	for (i = 0; i < 4*N + 6*nx_all; i++)
    	if (u[i] != u_ture[i])
    		printf("Wrong! \n");

    // reset
  	printf("Test GPU code. \n");
    for (i = 0; i < 4*N + 6*nx_all; i++) {
    	u[i] = cos((float)i/float(4*N + 6*nx_all));
    }  

    // print_array(g, N);
    // 
    float *dev_u, *dev_v, *dev_f, *dev_g, *dev_scratch;
    printf("N = %d \n", N);
    hipMalloc( (void**)&dev_u, N );
    hipMalloc( (void**)&dev_v, N );
    hipMalloc( (void**)&dev_f, N );
    hipMalloc( (void**)&dev_g, N );
    hipMalloc( (void**)&dev_scratch, 6*nx_all*sizeof(float) );

    hipMemcpy( dev_u, u, N, hipMemcpyHostToDevice);
    hipMemcpy( dev_v, v, N, hipMemcpyHostToDevice);
    hipMemcpy( dev_f, f, N, hipMemcpyHostToDevice);
    hipMemcpy( dev_g, g, N, hipMemcpyHostToDevice);
    hipMemcpy( dev_scratch, scratch, 
      6*nx_all*sizeof(float), 
      hipMemcpyHostToDevice
    );

    float *dev_dtcdx2, *dev_dtcdy2;
    int *dev_nx, *dev_ny;
    hipMalloc( (void**)&dev_dtcdx2, sizeof(float) );
    hipMalloc( (void**)&dev_dtcdy2, sizeof(float) ); 
    hipMalloc( (void**)&dev_nx, sizeof(int) );
    hipMalloc( (void**)&dev_ny, sizeof(int) );

    gpuErrchk(hipMemcpy(dev_dtcdx2, &dtcdx2, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_dtcdy2, &dtcdy2, sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_nx, &nx_all, sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_ny, &ny_all, sizeof(int), hipMemcpyHostToDevice));
    
	print_array(dev_u, N);
    central2d_predict_wrapper(
    		dev_v,
    		dev_scratch,
    		dev_u,
    		dev_f,
    		dev_g,
    		dev_dtcdx2,dev_dtcdy2,
            dev_nx,dev_ny,
            nfield, nx_all, ny_all // CPU
    );
    hipMemcpy( u, dev_u, N, hipMemcpyDeviceToHost);
    hipMemcpy( v, dev_v, N, hipMemcpyDeviceToHost);
    hipMemcpy( scratch, dev_scratch, 6*nx_all*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( f, dev_f, N, hipMemcpyDeviceToHost);
    hipMemcpy( g, dev_g, N, hipMemcpyDeviceToHost);

    // print_array(g, N);
   	printf("Check correctness\n");
	for (i = 0; i < 4*N + 6*nx_all; i++) {
    	if (u[i] != u_ture[i]) {
    		printf("Wrong! %f >>><<<< %f \n", u[i], u_ture[i]);
    	}
    }

}